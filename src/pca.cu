#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include "hipblas.h"
#include "include/pca.cuh"
#include "include/svd.cuh"
#include "include/cycleTimer.h"
#include <hip/hip_runtime.h>
#ifndef min
#define min(a, b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a, b) ((a > b) ? a : b)
#endif
//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
  printf("Cuda assert no error\n");
}
#else
#define cudaCheckError(ans) ans
#endif
#define DEBUG
#ifdef DEBUG
#define cublasCheckError(ans) cublasAssert((ans), __FILE__, __LINE__);
inline void cublasAssert(hipblasStatus_t code, const char *file, int line,
                         bool abort = true) {
  if (code != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS Error: %d at %s:%d\n", code, file, line);
    if (abort)
      exit(code);
  }
  printf("Cuda assert no error\n");
}
#else
#define cudaCheckError(ans) ans
#endif

__global__ void get_average_from_total(float *total, int n, int m) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < n) {
    total[row] = total[row] / m;
  }
  __syncthreads();
}

__global__ void subtract(float *matrix, float *averages, int m, int n) {
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < n && row < m) {
    matrix[row * n + col] = matrix[row * n + col] - averages[col];
  }
  __syncthreads();
}

__global__ void mult_S_U(float *out, float *S, float *U, int features,
                         int samples, int k) {
  // S is a diagonal matrix represented as a vector
  // Note col and row switched since we are dealing with row col order
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < k && row < samples) {
    out[row * k + col] = S[col] * U[col * samples + row];
  }
  __syncthreads();
}

void print_cpu_matrix(int m, int n, const float *A, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[col + row * n];
      printf("(%d,%d)%.3f,", row, col, Areg);
    }
    printf("\n");
  }
}

void printColMatrix(int m, int n, const float *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[row + col * lda];
      printf("%s(%d,%d) = %.3f\n", name, row + 1, col + 1, Areg);
    }
  }
}

void print_host_matrix(int m, int n, const float *A, const char *name) {
  float *tempmatrix;
  tempmatrix = (float *)malloc(sizeof(float) * m * n);
  hipMemcpy(tempmatrix, A, sizeof(float) * m * n, hipMemcpyHostToDevice);
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = tempmatrix[col + row * n];
      printf("(%d,%d)%.3f,", row, col, Areg);
    }
    printf("\n");
  }
}

float *transform(int nsamples, int nfeatures, int ncomponents, svd_t svd) {
  hipblasHandle_t handle;
  float alpha = 1.0;
  float beta = 0.0;
  hipblasOperation_t transa = HIPBLAS_OP_N; // no transpose
  hipblasOperation_t transb = HIPBLAS_OP_N; // no transpose
  float *out_mat = (float *)malloc(sizeof(float) * nsamples * nfeatures);
  for (int i = 0; i < nsamples * nfeatures; i++) {
    out_mat[i] = 0.0f;
  }
  float *d_out_mat = NULL;

  hipMalloc((void **)&d_out_mat, sizeof(float) * nsamples * nfeatures);
  hipMemcpy(d_out_mat, out_mat, sizeof(float) * nsamples * nfeatures,
             hipMemcpyHostToDevice);
  cublasCheckError(hipblasCreate(&handle));
  // assert(hipSuccess == cudaStat1);
  cublasCheckError(hipblasSgemm(handle, transa, transb, nsamples, nfeatures,
                               nfeatures, &alpha, svd.U, nsamples, svd.S,
                               nfeatures, &beta, d_out_mat, nsamples));

  hipMemcpy(out_mat, d_out_mat, sizeof(float) * nsamples * nfeatures,
             hipMemcpyDeviceToHost);
  if (d_out_mat)
    hipFree(d_out_mat);
  // printMatrix(nsamples, nfeatures, out_mat, nsamples, "transformed matrix");
  // print_cpu_matrix(nsamples, nfeatures, out_mat, "transformed matrix");
  cublasCheckError(hipblasDestroy(handle));
  return out_mat;
}

float *mean_shift(float *matrix, int M, int N) {
  hipblasHandle_t handle;
  float *x = new float[M];
  float *y = new float[N];
  float *d_matrix = NULL;
  float *clonem = NULL;
  float *d_x = NULL;
  float *d_y = NULL;
  float alpha = 1.0;
  float beta = 0.0;
  for (int i = 0; i < M; i++) {
    x[i] = 1.0f;
  }
  for (int i = 0; i < N; i++) {
    y[i] = 0.0f;
  }
  cudaCheckError(hipDeviceSynchronize());
  // will need to call hipblasDestroy() at some point
  cublasCheckError(hipblasCreate(&handle));
  cudaCheckError(hipDeviceSynchronize());
  cudaCheckError(hipMalloc((void **)&d_matrix, M * N * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&clonem, M * N * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_x, M * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_y, N * sizeof(float)));

  cudaCheckError(hipMemcpy(d_x, x, M * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                            hipMemcpyHostToDevice));


  cudaCheckError(hipDeviceSynchronize());
  // or HIPBLAS_OP_T?
  cublasCheckError(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, d_matrix, N,
                               d_x, 1, &beta, d_y, 1));

  cudaCheckError(hipDeviceSynchronize());
  const int threadsPerBlock = 512;
  int blocks = N / threadsPerBlock;
  if (N % threadsPerBlock != 0) {
    blocks++;
  }
  int LBLK = 32;
  dim3 tpb(LBLK, LBLK);
  int div = N / LBLK;
  int div2 = M / LBLK;
  if (N % LBLK != 0) {
    div++;
  }
  if (M % LBLK != 0) {
    div2++;
  }
  dim3 bs(div2, div);
  get_average_from_total<<<blocks, threadsPerBlock>>>(d_y, N, M);
  cudaCheckError(hipDeviceSynchronize());
  subtract<<<bs, tpb>>>(d_matrix, d_y, M, N);
  cudaCheckError(hipDeviceSynchronize());

  cublasCheckError(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, &alpha,
                               d_matrix, N, &beta, d_matrix, M, clonem, M));
  if (d_y)
    cudaCheckError(hipFree(d_y));
  if (d_x)
    cudaCheckError(hipFree(d_x));
  if (d_matrix)
    cudaCheckError(hipFree(d_matrix));
  if (x)
    free(x);
  if (y)
    free(y);
  return clonem;
}

float *pca_from_S_U(svd_t svd, int M, int N, int k) {
  float *out = NULL;
  float *out_cpu = NULL;
  // Create out matrix
  out_cpu = (float *)malloc(k * M * sizeof(float));
  cudaCheckError(hipMalloc((void **)&out, k * M * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&out, k * M * sizeof(float)));
  // Create kernel parameters
  int LBLK = 32;
  dim3 tpb(LBLK, LBLK);
  int div = k / LBLK;
  int div2 = M / LBLK;
  if (k % LBLK != 0) {
    div++;
  }
  if (M % LBLK != 0) {
    div2++;
  }
  dim3 bs(div2, div);
  // Call kernel
  mult_S_U<<<bs, tpb>>>(out, svd.S, svd.U, N, M, k);
  cudaCheckError(hipFree(svd.V));
  cudaCheckError(hipFree(svd.U));
  cudaCheckError(hipFree(svd.S));
  cudaCheckError(
      hipMemcpy(out_cpu, out, k * M * sizeof(float), hipMemcpyDeviceToHost));
  cudaCheckError(hipDeviceSynchronize());
  return out_cpu;
}

float_matrix_t perform_pca(float *matrix, int M, int N, int ncomponents, const int econ, const float tol, 
                            const int max_sweeps, const bool verbose, TimeLogger* tl) {
  TimeLogger::timeLog* mean_shift_log;
  TimeLogger::timeLog* perform_svd_log;
  TimeLogger::timeLog* memcpy_log;
  TimeLogger::timeLog* pca_S_U_log;

  if(tl != NULL) 
    mean_shift_log = tl->start("mean_shift()");
  float *d_matrix = mean_shift(matrix, M, N);
  if(tl != NULL) {
    cudaCheckError(hipDeviceSynchronize());
    tl->stop(mean_shift_log);
    perform_svd_log = tl->start("perform_svd()");
  }
  double startTime = CycleTimer::currentSeconds();
  svd_t svd =
      perform_svd(d_matrix, M, N, econ, tol, max_sweeps, verbose);

  double endTime = CycleTimer::currentSeconds();
  printf("%.2f ms\n", 1000.f * (endTime - startTime));
  float_matrix_t svd_out;
  if(tl != NULL) {
    cudaCheckError(hipDeviceSynchronize());
    tl->stop(perform_svd_log);
    memcpy_log = tl->start("svd matrices to device memory");
  }
  int minmn = min(M, N);
  svd_out.S = (float *) malloc(sizeof(float) * minmn);
  svd_out.V = (float *) malloc(sizeof(float) * N * N);
  cudaCheckError(
    hipMemcpy(svd_out.V, svd.V, N * N * sizeof(float), hipMemcpyDeviceToHost));
  cudaCheckError(
    hipMemcpy(svd_out.S, svd.S, minmn * sizeof(float), hipMemcpyDeviceToHost));
  cudaCheckError(hipDeviceSynchronize());
  if(tl != NULL) {
    tl->stop(memcpy_log);
    pca_S_U_log = tl->start("pca_from_S_U");
  }
  svd_out.matrix = pca_from_S_U(svd, M, N, ncomponents);
  svd_out.rows = M;
  svd_out.cols = ncomponents;
  cudaCheckError(hipDeviceSynchronize());
  if(tl != NULL) tl->stop(pca_S_U_log);
  return svd_out;
}