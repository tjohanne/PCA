#include "hip/hip_runtime.h"
#include "include/svd.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#ifndef min
#define min(a, b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a, b) ((a > b) ? a : b)
#endif
#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif
#define DEBUG
#ifdef DEBUG
#define cublasCheckError(ans) cublasAssert((ans), __FILE__, __LINE__);
inline void cublasAssert(hipblasStatus_t code, const char *file, int line,
                         bool abort = true) {
  if (code != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS Error: %d at %s:%d\n", code, file, line);
    if (abort)
      exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif




/**
 * @brief 
 * 
 * @param m 
 * @param n 
 * @param A 
 * @param lda 
 * @param name 
 */
void printMatrix(int m, int n, const float *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[row + col * lda];
      printf("%s(%d,%d) = %.3f\n", name, row + 1, col + 1, Areg);
    }
  }
}



/**
 * @brief 
 * 
 * @param m 
 * @param n 
 * @param A 
 * @param lda 
 * @param name 
 */
void printMatrixcsv(int m, int n, const float *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[row + col * lda];
      printf("%.3f,", Areg);
    }
    printf("\n");
  }
}

void printVector(int m, const float *A, const char *name) {
  for (int i = 0; i < m; i++) {
    float Areg = A[i];
    printf("%.6f\n", Areg);
    printf("%s(%d) = %.3f\n", name, i, Areg);
  }
}


/**
 * @brief 
 * 
 * @param d_A 
 * @param m 
 * @param n 
 * @param economy 
 * @param tolerance 
 * @param max_sweeps 
 * @param verbose 
 * @return svd_t 
 */
svd_t perform_svd(float *d_A, int m, int n, int economy, const float tolerance,
                  const int max_sweeps, bool verbose) {
  hipsolverHandle_t cusolverH = NULL;
  hipStream_t stream = NULL;
  hipsolverGesvdjInfo_t gesvdj_params = NULL;
  hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
  const int lda = m;
  const int ldu = m;
  const int ldv = n;
  const int minmn = min(m, n);
  float *d_S = NULL;
  float *d_U = NULL;
  float *d_V = NULL;
  int *d_info = NULL;   /* error info */
  int lwork = 0;        /* size of workspace */
  float *d_work = NULL; /* devie workspace for gesvdj */
  int info = 0;         /* host copy of error info */
  const hipsolverEigMode_t jobz =
      HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
  double residual = 0;
  int executed_sweeps = 0;
  /* create cusolver handle */
  status = hipsolverDnCreate(&cusolverH);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  cudaCheckError(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  status = hipsolverSetStream(cusolverH, stream);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  status = hipsolverDnXgesvdjSetTolerance(gesvdj_params, tolerance);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  status = hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  cudaCheckError(hipMalloc((void **)&d_S, sizeof(float) * minmn));
  cudaCheckError(hipMalloc((void **)&d_U, sizeof(float) * ldu * m));
  cudaCheckError(hipMalloc((void **)&d_V, sizeof(float) * ldv * n));
  cudaCheckError(hipMalloc((void **)&d_info, sizeof(int)));
  status = hipsolverDnSgesvdj_bufferSize(cusolverH, jobz, economy,
                                        m, //  nrows
                                        n, //  ncols
                                        d_A, lda, d_S, d_U, ldu, d_V, ldv,
                                        &lwork, gesvdj_params);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  cudaCheckError(hipMalloc((void **)&d_work, sizeof(float) * lwork));

  /* compute SVD */
  status =
      hipsolverDnSgesvdj(cusolverH, jobz, economy, m, n, d_A, lda, d_S, d_U, ldu,
                        d_V, ldv, d_work, lwork, d_info, gesvdj_params);
  cudaCheckError(hipDeviceSynchronize());
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  cudaCheckError(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
  cudaCheckError(hipDeviceSynchronize());

  if (0 == info) {
    printf("gesvdj converges \n");
  } else if (0 > info) {
    printf("%d-th parameter is wrong \n", -info);
    exit(1);
  } else {
    printf("WARNING: info = %d : gesvdj does not converge \n", info);
  }

  if (verbose) {

    status =
        hipsolverDnXgesvdjGetSweeps(cusolverH, gesvdj_params, &executed_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    status = hipsolverDnXgesvdjGetResidual(cusolverH, gesvdj_params, &residual);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    printf("residual |A - U*S*V**H|_F = %E \n", residual);
    printf("number of executed sweeps = %d \n", executed_sweeps);
  }
  /*  free resources  */
  if (d_A)
    hipFree(d_A);
  if (d_info)
    hipFree(d_info);
  if (d_work)
    hipFree(d_work);
  if (cusolverH)
    hipsolverDnDestroy(cusolverH);
  if (stream)
    hipStreamDestroy(stream);
  if (gesvdj_params)
    hipsolverDnDestroyGesvdjInfo(gesvdj_params);
  SVD svd;
  svd.S = d_S;
  svd.V = d_V;
  svd.U = d_U;
  return svd;
}
