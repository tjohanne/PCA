#include "hip/hip_runtime.h"
#include "include/stats.cuh"
#include "include/gpuasserts.cuh"
#include "debugFunctions.cu"
/**
 * @brief Takes a matrix thats been summed r
 * 
 * @param total 
 * @param n 
 * @param m 
 * @return __global__ 
 */
__global__ void get_average_from_total(float *total, int n, int m) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < n) {
    total[row] = total[row] / m;
  }
  __syncthreads();
}

/**
 * @brief subtracts average vector from each matrix entry 
 * 
 * @param matrix matrix of size m x n
 * @param averages 
 * @param m 
 * @param n 
 */
__global__ void subtract(float *matrix, float *averages, int m, int n) {
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < n && row < m) {
    matrix[row * n + col] = matrix[row * n + col] - averages[col];
  }
  __syncthreads();
}

float* row_to_column_order(float *d_matrix, int M, int N, int batch_size, hipblasHandle_t handle) {
  float alpha = 1.0;
  float beta = 0.0;
  float *clonem = NULL;
  cudaCheckError(hipMalloc((void **)&clonem, M * N * sizeof(float)));
  int m = M / batch_size;
  int stride = m * N;
  float* clonenext = clonem;
  for(int i = 0; i < batch_size; i++) {
    cublasCheckError(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, N, &alpha,
                               d_matrix, N, &beta, d_matrix, M, clonenext, m));
    clonenext += stride;
    d_matrix += stride;
  }
  return clonem;
}

/**
 * @brief Centers the original input matrix by computing 
 * the mean for each feature, and subtracting the mean 
 * from each observation.
 * 
 * @param matrix 
 * @param M 
 * @param N 
 * @return float* 
 */
float *mean_shift(float *matrix, int M, int N, int batch_size, hipblasHandle_t handle) {
  float *x = new float[M];
  float *y = new float[N];
  float *d_matrix = NULL;
  float *d_x = NULL;
  float *d_y = NULL;
  float alpha = 1.0;
  float beta = 0.0;
  for (int i = 0; i < M; i++) {
    x[i] = 1.0f;
  }
  for (int i = 0; i < N; i++) {
    y[i] = 0.0f;
  }
  cudaCheckError(hipMalloc((void **)&d_matrix, M * N * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_x, M * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_y, N * sizeof(float)));

  cudaCheckError(hipMemcpy(d_x, x, M * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                            hipMemcpyHostToDevice));


  cudaCheckError(hipDeviceSynchronize());
  cublasCheckError(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, d_matrix, N,
                               d_x, 1, &beta, d_y, 1));

  cudaCheckError(hipDeviceSynchronize());
  const int threadsPerBlock = 512;
  int blocks = N / threadsPerBlock;
  if (N % threadsPerBlock != 0) {
    blocks++;
  }
  int LBLK = 32;
  dim3 tpb(LBLK, LBLK);
  int div = N / LBLK;
  int div2 = M / LBLK;
  if (N % LBLK != 0) {
    div++;
  }
  if (M % LBLK != 0) {
    div2++;
  }
  dim3 bs(div2, div);
  get_average_from_total<<<blocks, threadsPerBlock>>>(d_y, N, M);
  cudaCheckError(hipDeviceSynchronize());
  subtract<<<bs, tpb>>>(d_matrix, d_y, M, N);
  cudaCheckError(hipDeviceSynchronize());

  float* clonem = row_to_column_order(d_matrix, M, N, batch_size, handle);  
  if (d_y)
    cudaCheckError(hipFree(d_y));
  if (d_x)
    cudaCheckError(hipFree(d_x));
  if (d_matrix)
    cudaCheckError(hipFree(d_matrix));
  if (x)
    free(x);
  if (y)
    free(y);
  return clonem;
}

int main() {
  float A[12] = {1.0, 2.0, 4.0, 5.0, 2.0, 1.0, 10.0, 9.0, 8.0, 7.0, 6.0, 5.0};
  float *a = new float[12];
  int mrows = 4;
  int ncols = 3;
  memcpy(a, A, 12 * sizeof(float));
  print_cpu_matrix(mrows, ncols, A);
  float *d_A = NULL;
  cudaCheckError(hipMalloc((void **)&d_A, 12 * sizeof(float)));
  cudaCheckError(hipMemcpy(d_A, a, 12 * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(hipDeviceSynchronize());

  hipsolverHandle_t cusolverH = NULL;
  hipblasHandle_t cublasH = NULL;
  cusolverCheckError(hipsolverDnCreate(&cusolverH));
  cublasCheckError(hipblasCreate(&cublasH));
  // print_device_vector(ncols * mrows, d_A);
  float* d_batch = row_to_column_order(d_A, mrows, ncols, 2, cublasH);
  float* d_norm = row_to_column_order(d_A, mrows, ncols, 1, cublasH);
  print_device_vector(ncols * mrows, d_A);
  printf("\n");
  print_device_vector(ncols * mrows, d_batch);
  print_device_vector(ncols * mrows, d_norm);
}