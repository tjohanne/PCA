#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "hipblas.h"
#include "svd.cuh"
#include <hip/hip_runtime.h>
#ifndef min
#define min(a, b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a, b) ((a > b) ? a : b)
#endif
//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//
#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
  printf("Cuda assert no error\n");
}
#else
#define cudaCheckError(ans) ans
#endif
#define DEBUG
#ifdef DEBUG
#define cublasCheckError(ans) cublasAssert((ans), __FILE__, __LINE__);
inline void cublasAssert(hipblasStatus_t code, const char *file, int line,
                         bool abort = true) {
  if (code != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS Error: %d at %s:%d\n", code, file, line);
    if (abort)
      exit(code);
  }
  printf("Cuda assert no error\n");
}
#else
#define cudaCheckError(ans) ans
#endif

__global__ void get_average_from_total(float *total, int n, int m) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < n) {
    total[row] = total[row] / m;
  }
  __syncthreads();
}

__global__ void subtract(float *matrix, float *averages, int m, int n) {
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < n && row < m) {
    matrix[row * n + col] = matrix[row * n + col] - averages[col];
  }
  __syncthreads();
}

void print_cpu_matrix(int m, int n, const float *A, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[col + row * n];
      printf("%.3f,", Areg);
    }
    printf("\n");
  }
}

float *mean_shift(float *matrix, int M, int N) {
  hipblasHandle_t handle;
  float *x = new float[M];
  float *y = new float[N];
  float *d_matrix = NULL;
  float *d_x = NULL;
  float *d_y = NULL;
  float alpha = 1.0;
  float beta = 0.0;
  // float *alpha = new float[1];
  // *alpha = 1.0f;
  // float *beta = new float[1];
  // *beta = 1.0f;
  printMatrix(M, N, matrix, M, "matrix");
  for (int i = 0; i < M; i++) {
    x[i] = 1.0f;
  }
  for (int i = 0; i < N; i++) {
    y[i] = 0.0f;
  }
  // will need to call hipblasDestroy() at some point
  cublasCheckError(hipblasCreate(&handle));
  cudaCheckError(hipMalloc((void **)&d_matrix, M * N * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_x, M * sizeof(float)));
  cudaCheckError(hipMalloc((void **)&d_y, N * sizeof(float)));
  cudaCheckError(hipMemcpy(d_x, x, M * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_matrix, matrix, M * N * sizeof(float),
                            hipMemcpyHostToDevice));
  // or HIPBLAS_OP_T?
  cublasCheckError(hipblasSgemv(handle, HIPBLAS_OP_N, N, M, &alpha, d_matrix, N,
                               d_x, 1, &beta, d_y, 1));
  const int threadsPerBlock = 512;
  int blocks = N / threadsPerBlock;
  if (N % threadsPerBlock != 0) {
    blocks++;
  }
  int LBLK = 32;
  dim3 tpb(LBLK, LBLK);
  int div = N / LBLK;
  int div2 = M / LBLK;
  if (N % LBLK != 0) {
    div++;
  }
  if (M % LBLK != 0) {
    div2++;
  }
  dim3 bs(div2, div);
  get_average_from_total<<<blocks, threadsPerBlock>>>(d_y, N, M);
  cudaCheckError(hipDeviceSynchronize());
  subtract<<<bs, tpb>>>(d_matrix, d_y, M, N);
  cudaCheckError(hipDeviceSynchronize());
  if (d_y)
    cudaCheckError(hipFree(d_y));
  if (d_x)
    cudaCheckError(hipFree(d_x));
  if (x)
    free(x);
  if (y)
    free(y);
  // print_cpu_matrix(M, N, matrix, "matrix");
  return d_matrix;
}

void perform_pca(float *matrix, int M, int N) {
  float *d_matrix = mean_shift(matrix, M, N);
  svd_t svd = perform_svd(d_matrix, M, N);
}