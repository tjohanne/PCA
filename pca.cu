#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "hipblas.h"
#include "svd.cuh"
#include <hip/hip_runtime.h>
#ifndef min
#define min(a, b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a, b) ((a > b) ? a : b)
#endif
//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//

void perform_pca(float *matrix, int M, int N) {
  matrix[0] = 4.0;
  matrix[1] = 0.0;
  matrix[2] = 3.0;
  matrix[3] = -5.0;
  svd_t svd = perform_svd(matrix, 2, 2);
}