#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include gesvdj_example.cpp
 *   g++ -o gesvdj_example gesvdj_example.o -L/usr/local/cuda/lib64 -lcudart
 * -lcusolver
 *   TODO check nvcc flag?
 */
#include "svd.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#ifndef min
#define min(a, b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a, b) ((a > b) ? a : b)
#endif
void printMatrix(int m, int n, const float *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[row + col * lda];
      printf("%s(%d,%d) = %.3f\n", name, row + 1, col + 1, Areg);
    }
  }
}

void printMatrixcsv(int m, int n, const float *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[row + col * lda];
      printf("%.3f,", Areg);
    }
    printf("\n");
  }
}

void printVector(int m, const float *A, const char *name) {
  for (int i = 0; i < m; i++) {
    float Areg = A[i];
    printf("%.6f\n", Areg);
    printf("%s(%d) = %.3f\n", name, i, Areg);
  }
}

__global__ void vec_to_diag(float *vec, float *diag_mat, int vec_length) {
  int diag_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (diag_index < vec_length) {
    diag_mat[vec_length * diag_index + diag_index] = vec[diag_index];
  }
  __syncthreads();
}

svd_t perform_svd(float *d_A, int m, int n, int economy, const float tolerance,
                  const int max_sweeps, bool verbose) {
  hipsolverHandle_t cusolverH = NULL;
  hipStream_t stream = NULL;
  hipsolverGesvdjInfo_t gesvdj_params = NULL;
  hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;
  hipError_t cudaStat4 = hipSuccess;
  hipError_t cudaStat5 = hipSuccess;
  const int lda = m;
  const int ldu = m;
  const int ldv = n;
  const int minmn = min(m, n);
  float *U = new float[ldu * m];
  float *V = new float[ldv * n];
  float *S = new float[minmn * minmn];
  float *d_S = NULL;
  float *d_U = NULL;
  float *d_V = NULL;
  int *d_info = NULL;   /* error info */
  int lwork = 0;        /* size of workspace */
  float *d_work = NULL; /* devie workspace for gesvdj */
  int info = 0;         /* host copy of error info */
  const hipsolverEigMode_t jobz =
      HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
  double residual = 0;
  int executed_sweeps = 0;

  /* create cusolver handle */
  status = hipsolverDnCreate(&cusolverH);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  assert(hipSuccess == cudaStat1);
  status = hipsolverSetStream(cusolverH, stream);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  status = hipsolverDnXgesvdjSetTolerance(gesvdj_params, tolerance);
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  status = hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  cudaStat2 = hipMalloc((void **)&d_S, sizeof(float) * minmn);
  cudaStat3 = hipMalloc((void **)&d_U, sizeof(float) * ldu * m);
  cudaStat4 = hipMalloc((void **)&d_V, sizeof(float) * ldv * n);
  cudaStat5 = hipMalloc((void **)&d_info, sizeof(int));
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);
  assert(hipSuccess == cudaStat4);
  assert(hipSuccess == cudaStat5);
  assert(hipSuccess == cudaStat5);

  status = hipsolverDnSgesvdj_bufferSize(cusolverH, jobz, economy,
                                        m, //  nrows
                                        n, //  ncols
                                        d_A, lda, d_S, d_U, ldu, d_V, ldv,
                                        &lwork, gesvdj_params);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  cudaStat1 = hipMalloc((void **)&d_work, sizeof(float) * lwork);
  assert(hipSuccess == cudaStat1);

  /* compute SVD */
  status =
      hipsolverDnSgesvdj(cusolverH, jobz, economy, m, n, d_A, lda, d_S, d_U, ldu,
                        d_V, ldv, d_work, lwork, d_info, gesvdj_params);
  cudaStat1 = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  assert(hipSuccess == cudaStat1);

  const int threadsPerBlock = 512;
  int blocks = minmn / threadsPerBlock;
  if (minmn % threadsPerBlock != 0) {
    blocks++;
  }

  cudaStat1 =
      hipMemcpy(U, d_U, sizeof(float) * ldu * m, hipMemcpyDeviceToHost);
  cudaStat2 =
      hipMemcpy(V, d_V, sizeof(float) * ldv * n, hipMemcpyDeviceToHost);
  cudaStat3 = hipMemcpy(S, d_S, sizeof(float) * minmn, hipMemcpyDeviceToHost);
  cudaStat4 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
  cudaStat5 = hipDeviceSynchronize();
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);
  assert(hipSuccess == cudaStat4);
  assert(hipSuccess == cudaStat5);

  if (0 == info) {
    printf("gesvdj converges \n");
  } else if (0 > info) {
    printf("%d-th parameter is wrong \n", -info);
    exit(1);
  } else {
    printf("WARNING: info = %d : gesvdj does not converge \n", info);
  }

  if (verbose) {
    printf("S = singular values (matlab base-1)\n");
    printMatrix(minmn, 1, S, minmn, "S");
    printf("=====\n");

    printf("U = left singular vectors (matlab base-1)\n");
    printMatrix(m, m, U, ldu, "U");
    printf("=====\n");

    printf("V = right singular vectors (matlab base-1)\n");
    printMatrix(n, n, V, ldv, "V");
    printf("=====\n");

    printf("S = matrix (matlab base-1)\n");
    printMatrix(minmn, 1, S, minmn, "S MATRIX");
    printf("=====\n");

    status =
        hipsolverDnXgesvdjGetSweeps(cusolverH, gesvdj_params, &executed_sweeps);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    status = hipsolverDnXgesvdjGetResidual(cusolverH, gesvdj_params, &residual);
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    printf("residual |A - U*S*V**H|_F = %E \n", residual);
    printf("number of executed sweeps = %d \n", executed_sweeps);
  }

  /*  free resources  */
  if (d_A)
    hipFree(d_A);
  // if (d_S)
  //   hipFree(d_S);
  //   if (d_U)
  //     hipFree(d_U);
  if (d_V)
    hipFree(d_V);
  if (d_info)
    hipFree(d_info);
  if (d_work)
    hipFree(d_work);
  if (U)
    free(U);
  if (V)
    free(V);
  if (S)
    free(S);
  if (cusolverH)
    hipsolverDnDestroy(cusolverH);
  if (stream)
    hipStreamDestroy(stream);
  if (gesvdj_params)
    hipsolverDnDestroyGesvdjInfo(gesvdj_params);
  SVD svd;
  svd.S = d_S;
  svd.V = d_V;
  svd.U = d_U;
  // hipDeviceReset();
  return svd;
}
