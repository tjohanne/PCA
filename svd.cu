#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include gesvdj_example.cpp
 *   g++ -o gesvdj_example gesvdj_example.o -L/usr/local/cuda/lib64 -lcudart
 * -lcusolver
 *   TODO check nvcc flag?
 */
#include "svd.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#ifndef min
#define min(a, b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a, b) ((a > b) ? a : b)
#endif
void printMatrix(int m, int n, const float *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[row + col * lda];
      printf("%s(%d,%d) = %.3f\n", name, row + 1, col + 1, Areg);
    }
  }
}

void printMatrixcsv(int m, int n, const float *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      float Areg = A[row + col * lda];
      printf("%.3f,", Areg);
    }
    printf("\n");
  }
}

void printVector(int m, const float *A, const char *name) {
  for (int i = 0; i < m; i++) {
    float Areg = A[i];
    printf("%.6f\n", Areg);
    printf("%s(%d) = %.3f\n", name, i, Areg);
  }
}

__global__ void vec_to_diag(float *vec, float *diag_mat, int vec_length) {
  int diag_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (diag_index < vec_length) {
    diag_mat[vec_length * diag_index + diag_index] = vec[diag_index];
  }
  __syncthreads();
}

svd_t perform_svd(float *d_A, int m, int n) {
  hipsolverHandle_t cusolverH = NULL;
  hipStream_t stream = NULL;
  hipsolverGesvdjInfo_t gesvdj_params = NULL;

  hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;
  hipError_t cudaStat4 = hipSuccess;
  hipError_t cudaStat5 = hipSuccess;
  const int lda = m; /* A is m-by-n */
  const int ldu = m; /* U is m-by-m */
  const int ldv = n; /* V is n-by-n */
  const int minmn = min(m, n);
  /*       | 1 2  |
   *   A = | 4 5  |
   *       | 2 1  |
   */
  //   float A[lda * n] = {4.0, 0.0, 3.0, -5.0};
  float *U = new float[ldu * m];
  float *V = new float[ldv * n];
  float *S = new float[minmn * minmn];
  // float U[ldu*m]; /* m-by-m unitary matrix, left singular vectors  */
  // float V[ldv*n]; /* n-by-n unitary matrix, right singular vectors */
  // float S[minmn];     /* numerical singular value */
  /* exact singular values */
  //  TODO s_exact is for testing, remove
  float S_exact[2 * 3] = {6.3, 3.16};
  //   float *d_A = NULL;    /* device copy of A */
  float *d_S = NULL; /* singular values */
  float *d_U = NULL; /* left singular vectors */
  float *d_V = NULL; /* right singular vectors */
  float *d_Smat = NULL;
  int *d_info = NULL;   /* error info */
  int lwork = 0;        /* size of workspace */
  float *d_work = NULL; /* devie workspace for gesvdj */
  int info = 0;         /* host copy of error info */
                        /* configuration of gesvdj  */
  const float tol = 1.e-7;
  const int max_sweeps = 15;
  const hipsolverEigMode_t jobz =
      HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
  const int econ = 0;           /* econ = 1 for economy size */

  /* numerical results of gesvdj  */
  double residual = 0;
  int executed_sweeps = 0;

  printf("example of gesvdj \n");
  printf("tol = %E, default value is machine zero \n", tol);
  printf("max. sweeps = %d, default value is 100\n", max_sweeps);
  printf("econ = %d \n", econ);

  printf("A = (matlab base-1)\n");
  //   printMatrix(m, n, A, lda, "A");
  printf("=====\n");

  /* step 1: create cusolver handle, bind a stream */
  status = hipsolverDnCreate(&cusolverH);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  assert(hipSuccess == cudaStat1);

  status = hipsolverSetStream(cusolverH, stream);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  /* step 2: configuration of gesvdj */
  status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  /* default value of tolerance is machine zero */
  status = hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  /* default value of max. sweeps is 100 */
  status = hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  /* step 3: copy A and B to device */
  // cudaStat1 = hipMalloc((void **)&d_A, sizeof(float) * lda * n);
  cudaStat2 = hipMalloc((void **)&d_S, sizeof(float) * minmn);
  cudaStat3 = hipMalloc((void **)&d_U, sizeof(float) * ldu * m);
  cudaStat4 = hipMalloc((void **)&d_V, sizeof(float) * ldv * n);
  cudaStat5 = hipMalloc((void **)&d_info, sizeof(int));
  cudaStat5 = hipMalloc((void **)&d_Smat, sizeof(float) * minmn * minmn);
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);
  assert(hipSuccess == cudaStat4);
  assert(hipSuccess == cudaStat5);
  assert(hipSuccess == cudaStat5);

  //   cudaStat1 =
  //   hipMemcpy(d_A, A, sizeof(float) * lda * n, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat1);
  /* step 4: query workspace of SVD */
  status = hipsolverDnSgesvdj_bufferSize(
      cusolverH,
      jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
      /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors
       */
      econ, /* econ = 1 for economy size */
      m,    /* nubmer of rows of A, 0 <= m */
      n,    /* number of columns of A, 0 <= n  */
      d_A,  /* m-by-n */
      lda,  /* leading dimension of A */
      d_S,  /* min(m,n) */
            /* the singular values in descending order */
      d_U,  /* m-by-m if econ = 0 */
            /* m-by-min(m,n) if econ = 1 */
      ldu,  /* leading dimension of U, ldu >= max(1,m) */
      d_V,  /* n-by-n if econ = 0  */
            /* n-by-min(m,n) if econ = 1  */
      ldv,  /* leading dimension of V, ldv >= max(1,n) */
      &lwork, gesvdj_params);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  cudaStat1 = hipMalloc((void **)&d_work, sizeof(float) * lwork);
  assert(hipSuccess == cudaStat1);

  /* step 5: compute SVD */
  status = hipsolverDnSgesvdj(
      cusolverH,
      jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
      /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors
       */
      econ, /* econ = 1 for economy size */
      m,    /* nubmer of rows of A, 0 <= m */
      n,    /* number of columns of A, 0 <= n  */
      d_A,  /* m-by-n */
      lda,  /* leading dimension of A */
      d_S,  /* min(m,n)  */
            /* the singular values in descending order */
      d_U,  /* m-by-m if econ = 0 */
            /* m-by-min(m,n) if econ = 1 */
      ldu,  /* leading dimension of U, ldu >= max(1,m) */
      d_V,  /* n-by-n if econ = 0  */
            /* n-by-min(m,n) if econ = 1  */
      ldv,  /* leading dimension of V, ldv >= max(1,n) */
      d_work, lwork, d_info, gesvdj_params);
  cudaStat1 = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == status);
  assert(hipSuccess == cudaStat1);

  const int threadsPerBlock = 64;
  int blocks = minmn / threadsPerBlock;
  if (minmn % threadsPerBlock != 0) {
    blocks++;
  }

  //  transform S from a vector to a diagonal matrix
  vec_to_diag<<<1, threadsPerBlock>>>(d_S, d_Smat, minmn);

  cudaStat1 =
      hipMemcpy(U, d_U, sizeof(float) * ldu * m, hipMemcpyDeviceToHost);
  cudaStat2 =
      hipMemcpy(V, d_V, sizeof(float) * ldv * n, hipMemcpyDeviceToHost);
  cudaStat3 = hipMemcpy(S, d_Smat, sizeof(float) * minmn * minmn,
                         hipMemcpyDeviceToHost);
  cudaStat4 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
  cudaStat5 = hipDeviceSynchronize();
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);
  assert(hipSuccess == cudaStat4);
  assert(hipSuccess == cudaStat5);

  if (0 == info) {
    printf("gesvdj converges \n");
  } else if (0 > info) {
    printf("%d-th parameter is wrong \n", -info);
    exit(1);
  } else {
    printf("WARNING: info = %d : gesvdj does not converge \n", info);
  }

  printf("S = singular values (matlab base-1)\n");
  printMatrix(minmn, 1, S, minmn, "S");
  printf("=====\n");

  printf("U = left singular vectors (matlab base-1)\n");
  printMatrix(m, m, U, ldu, "U");
  printf("=====\n");

  printf("V = right singular vectors (matlab base-1)\n");
  printMatrix(n, n, V, ldv, "V");
  printf("=====\n");

  printf("S = matrix (matlab base-1)\n");
  printMatrix(minmn, minmn, S, minmn, "S MATRIX");
  printf("=====\n");

  /* step 6: measure error of singular value */
  float ds_sup = 0;
  for (int j = 0; j < minmn; j++) {
    float err = fabs(S[j] - S_exact[j]);
    ds_sup = (ds_sup > err) ? ds_sup : err;
  }
  printf("|S - S_exact|_sup = %E \n", ds_sup);

  status =
      hipsolverDnXgesvdjGetSweeps(cusolverH, gesvdj_params, &executed_sweeps);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  status = hipsolverDnXgesvdjGetResidual(cusolverH, gesvdj_params, &residual);
  assert(HIPSOLVER_STATUS_SUCCESS == status);

  printf("residual |A - U*S*V**H|_F = %E \n", residual);
  printf("number of executed sweeps = %d \n", executed_sweeps);

  /*  free resources  */
  if (d_A)
    hipFree(d_A);
  if (d_S)
    hipFree(d_S);
  //   if (d_U)
  //     hipFree(d_U);
  //   if (d_V)
  //     hipFree(d_V);
  if (d_info)
    hipFree(d_info);
  if (d_work)
    hipFree(d_work);
  if (U)
    free(U);
  if (V)
    free(V);
  if (S)
    free(S);
  if (cusolverH)
    hipsolverDnDestroy(cusolverH);
  if (stream)
    hipStreamDestroy(stream);
  if (gesvdj_params)
    hipsolverDnDestroyGesvdjInfo(gesvdj_params);
  SVD svd;
  svd.S = d_Smat;
  svd.V = d_V;
  svd.U = d_U;
  // hipDeviceReset();
  return svd;
}
