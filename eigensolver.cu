/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include syevd_example.cpp
 *   g++ -o a.out syevd_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 *
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdlib.h>

void printMatrix(int m, int n, const double *A, int lda, const char *name) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < n; col++) {
      double Areg = A[row + col * lda];
      printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
    }
  }
}

int eigensolver_example() {
  hipsolverHandle_t cusolverH = NULL;
  hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
  hipError_t cudaStat1 = hipSuccess;
  hipError_t cudaStat2 = hipSuccess;
  hipError_t cudaStat3 = hipSuccess;
  const int m = 3;
  const int lda = m;
  /*       | 3.5 0.5 0 |
   *   A = | 0.5 3.5 0 |
   *       | 0   0   2 |
   *
   */
  double A[lda * m] = {3.5, 0.5, 0, 0.5, 3.5, 0, 0, 0, 2.0};
  double lambda[m] = {2.0, 3.0, 4.0};

  double V[lda * m]; // eigenvectors
  double W[m];       // eigenvalues

  double *d_A = NULL;
  double *d_W = NULL;
  int *devInfo = NULL;
  double *d_work = NULL;
  int lwork = 0;

  int info_gpu = 0;

  printf("A = (matlab base-1)\n");
  printMatrix(m, m, A, lda, "A");
  printf("=====\n");

  // step 1: create cusolver/cublas handle
  cusolver_status = hipsolverDnCreate(&cusolverH);
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

  // step 2: copy A and B to device
  cudaStat1 = hipMalloc((void **)&d_A, sizeof(double) * lda * m);
  cudaStat2 = hipMalloc((void **)&d_W, sizeof(double) * m);
  cudaStat3 = hipMalloc((void **)&devInfo, sizeof(int));
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);

  cudaStat1 =
      hipMemcpy(d_A, A, sizeof(double) * lda * m, hipMemcpyHostToDevice);
  assert(hipSuccess == cudaStat1);

  // step 3: query working space of syevd
  hipsolverEigMode_t jobz =
      HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  cusolver_status = hipsolverDnDsyevd_bufferSize(cusolverH, jobz, uplo, m, d_A,
                                                lda, d_W, &lwork);
  assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

  cudaStat1 = hipMalloc((void **)&d_work, sizeof(double) * lwork);
  assert(hipSuccess == cudaStat1);

  // step 4: compute spectrum
  cusolver_status = hipsolverDnDsyevd(cusolverH, jobz, uplo, m, d_A, lda, d_W,
                                     d_work, lwork, devInfo);
  cudaStat1 = hipDeviceSynchronize();
  assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
  assert(hipSuccess == cudaStat1);

  cudaStat1 = hipMemcpy(W, d_W, sizeof(double) * m, hipMemcpyDeviceToHost);
  cudaStat2 =
      hipMemcpy(V, d_A, sizeof(double) * lda * m, hipMemcpyDeviceToHost);
  cudaStat3 =
      hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
  assert(hipSuccess == cudaStat1);
  assert(hipSuccess == cudaStat2);
  assert(hipSuccess == cudaStat3);

  printf("after syevd: info_gpu = %d\n", info_gpu);
  assert(0 == info_gpu);

  printf("eigenvalue = (matlab base-1), ascending order\n");
  for (int i = 0; i < m; i++) {
    printf("W[%d] = %E\n", i + 1, W[i]);
  }

  printf("V = (matlab base-1)\n");
  printMatrix(m, m, V, lda, "V");
  printf("=====\n");

  // step 4: check eigenvalues
  double lambda_sup = 0;
  for (int i = 0; i < m; i++) {
    double error = fabs(lambda[i] - W[i]);
    lambda_sup = (lambda_sup > error) ? lambda_sup : error;
  }
  printf("|lambda - W| = %E\n", lambda_sup);

  // free resources
  if (d_A)
    hipFree(d_A);
  if (d_W)
    hipFree(d_W);
  if (devInfo)
    hipFree(devInfo);
  if (d_work)
    hipFree(d_work);

  if (cusolverH)
    hipsolverDnDestroy(cusolverH);

  hipDeviceReset();

  return 0;
}
